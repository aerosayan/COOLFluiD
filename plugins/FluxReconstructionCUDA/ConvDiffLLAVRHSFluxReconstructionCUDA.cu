#include "hip/hip_runtime.h"
#include "FluxReconstructionCUDA/ConvDiffLLAVRHSFluxReconstructionCUDA.hh"
#include "Framework/MeshData.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"

#include "FluxReconstructionMethod/FluxData.hh"
#include "FluxReconstructionMethod/KernelData.hh"
#include "FluxReconstructionMethod/CellData.hh"

#include "FluxReconstructionCUDA/FluxReconstructionCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "Framework/MathTypes.hh"
#include "NavierStokes/Euler2DVarSetT.hh"
#include "NavierStokes/Euler2DConsT.hh"
#include "NavierStokes/NavierStokes2DVarSetT.hh"
#include "NavierStokes/NavierStokes2DConsT.hh"
#include "NavierStokes/NSVarSetListT.hh"

#include "FluxReconstructionMethod/LaxFriedrichsFlux.hh"
#include <stdio.h>

//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::NavierStokes;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace FluxReconstructionMethod {

//////////////////////////////////////////////////////////////////////////////

#define FR_NSLLAV_RHS_PROV(__dim__,__svars__,__uvars__,__order__,__nbBThreads__,__providerName__) \
MethodCommandProvider<ConvDiffLLAVRHSFluxReconstructionCUDA<LaxFriedrichsFlux, \
                      VarSetListT<Euler##__dim__##__svars__##T, Euler##__dim__##__uvars__##T>,NSVarSetListT<NavierStokes##__dim__##__svars__##T, NavierStokes##__dim__##__uvars__##T>,__order__,__nbBThreads__>, \
		      FluxReconstructionSolverData,FluxReconstructionCUDAModule>	\
FR_RhsNSLLAV##__dim__##__svars__##__uvars__##__order__##__nbBThreads__##Provider(__providerName__);
// 48 block threads (default)
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 0, 48, "NSLLAVFRLaxFriedrichs2DConsP0")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 1, 48, "NSLLAVFRLaxFriedrichs2DConsP1")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 2, 48, "NSLLAVFRLaxFriedrichs2DConsP2")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 3, 48, "NSLLAVFRLaxFriedrichs2DConsP3")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 4, 48, "NSLLAVFRLaxFriedrichs2DConsP4")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 5, 48, "NSLLAVFRLaxFriedrichs2DConsP5")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 6, 48, "NSLLAVFRLaxFriedrichs2DConsP6")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 7, 48, "NSLLAVFRLaxFriedrichs2DConsP7")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 8, 48, "NSLLAVFRLaxFriedrichs2DConsP8")
FR_NSLLAV_RHS_PROV(2D, Cons, Cons, 9, 48, "NSLLAVFRLaxFriedrichs2DConsP9")
//FR_EULER_RHS_PROV(3D, Cons, Cons, 48, "EulerFRLaxFried3DCons")
//FR_NS_RHS_PROV(2D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD2DPrim")
//FR_NS_RHS_PROV(3D, ProjectionCons, ProjectionPrim, 48, "CellLaxFriedMHD3DPrim")
#undef FR_NS_RHS_PROV

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr)
{
  // copy the state node data to shared memory
  //for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, CFuint ORDER>
HOST_DEVICE void setFluxData(const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS,ORDER>* fd, const CFuint iSol)
{
  fd->setStateID(LEFT, stateID);
  CFreal* statePtrR = &kd->states[stateID*PHYS::NBEQS];  

  setState<PHYS>(fd->getState(iSol), statePtrR);

  fd->setNbSolPnts(kd->nbSolPnts);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  CFLog(INFO, name << " = \t");
  for (CFuint i = 0; i < SIZE; ++i) {
    CFLog(INFO, array[i] << " ");
  }
  CFLog(INFO, "\n");
}

//////////////////////////////////////////////////////////////////////////////

//template <typename MODEL>
//HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
//				     const CFreal* nodes, CFreal* midFaceCoord)
//{  
//  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
//  coord = 0.;
//  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
//  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
//  for (CFuint n = 0; n < nbFaceNodes; ++n) {
//    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
//    const CFuint nodeID = cell->getNodeID(faceIdx,n);
//    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
//    for (CFuint d = 0; d < MODEL::DIM; ++d) {
//      coord[d] += faceNode[d];
//    }
//  }
//  coord *= ovNbFaceNodes;
//}

//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC>
//__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				       const CFuint nbCells,
//				       CFreal* states, 
//				       CFreal* nodes,
//				       CFreal* centerNodes,
//				       CFreal* ghostStates,
//				       CFreal* ghostNodes,
//				       CFreal* uX,
//				       CFreal* uY,
//				       CFreal* uZ,
//				       CFreal* limiter,
//				       CFreal* updateCoeff, 
//				       CFreal* rhs,
//				       CFreal* normals,
//				       CFint* isOutward,
//				       const CFuint* cellInfo,
//				       const CFuint* cellStencil,
//				       const CFuint* cellFaces,
//				       const CFuint* cellNodes,
//				       const CFint*  neighborTypes,
//				       const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    
//    // compute and store cell gradients at once 
//    POLYREC polyRec(dcor);
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
//  }
//}
      
//////////////////////////////////////////////////////////////////////////////

//template <typename PHYS, typename POLYREC, typename LIMITER>
//__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
//				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
//				     const CFuint nbCells,
//				     CFreal* states, 
//				     CFreal* nodes,
//				     CFreal* centerNodes,
//				     CFreal* ghostStates,
//				     CFreal* ghostNodes,
//				     CFreal* uX,
//				     CFreal* uY,
//				     CFreal* uZ,
//				     CFreal* limiter,
//				     CFreal* updateCoeff, 
//				     CFreal* rhs,
//				     CFreal* normals,
//				     CFint* isOutward,
//				     const CFuint* cellInfo,
//				     const CFuint* cellStencil,
//				     const CFuint* cellFaces,
//				     const CFuint* cellNodes,
//				     const CFint*  neighborTypes,
//				     const Framework::CellConn* cellConn)
//{    
//  // each thread takes care of computing the gradient for one single cell
//  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
//  
//  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
//  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
//  // dcol->init(gdcol);
//  
//  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
//  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
//  // dcor->init(gdcor);
//  
//  if (cellID < nbCells) {    
//    // compute all cell quadrature points at once (size of this array is overestimated)
//    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
//    
//    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
//    CellData::Itr cell = cells.getItr(cellID);
//    const CFuint nbFacesInCell = cell.getNbFacesInCell();
//    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
//      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
//    }
//    
//    // compute cell-based limiter at once
//    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
//			   rhs, normals, uX, uY, uZ, isOutward);
//    LIMITER limt(dcol);
//    
//    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
//      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
//    }
//    else {
//      if (!dcor->freezeLimiter) {
//	// historical modification of the limiter
//	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
//	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
//	CFuint currID = cellID*PHYS::NBEQS;
//	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
//	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
//	}
//      }
//    }
//  }
//}
  
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYS, typename PHYSNS, CFuint ORDER>
__global__ void computeStateLocalRHSKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
                                  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                  typename PHYSNS::DTERM::template DeviceConfigOptions<NOTYPE>* dcopNS,
                                  typename PHYSNS::PTERM::template DeviceConfigOptions<NOTYPE>* dcopNSConv,
                                  const CFuint nbCells,
                                  const CFreal resFactor,
				  CFreal* states, 
                                  CFreal* gradients,
                                  CFreal* gradientsAV,
                                  CFreal* updateCoeff, 
				  CFreal* rhs,
                                  CFreal* solPntNormals,
                                  CFreal* flxPntNormals,
                                  CFreal* cellVolumes,
                                  CFint* faceDir,
                                  const CFuint nbSolPnts,
                                  const CFuint nbrFaces,
                                  const CFuint* faceFlxPntConn,
                                  const CFuint* stateIDs,
                                  const CFint* neighbCellIDs,
                                  const CFuint* neighbFaceIDs,
                                  const CFuint* innerCellIsLeft,
                                  const CFuint nbrFlxPnts,
                                  const CFuint nbrSolSolDep,
                                  const CFuint* solSolDep,
                                  const CFuint nbrSolFlxDep,
                                  const CFuint* solFlxDep,
                                  const CFuint nbrFlxSolDep,
                                  const CFuint* flxSolDep,
                                  const CFreal* solPolyDerivAtSolPnts,
                                  const CFreal* solPolyValsAtFlxPnts,
                                  const CFuint* flxPntFlxDim,
                                  const CFreal* corrFctDiv,
                                  const CFreal* faceIntCoeff,
                                  const CFreal cflConvDiffRatio,
                                  const CFuint* nbNodeNeighbors,
                                  const CFreal* nodeEpsilons,
                                  const CFuint nbrCornerNodes,
                                  const CFuint* neighbNodeIDs,
                                  const CFuint* faceNeighbNodeIDs,
                                  const CFuint nbFaceNodes,
                                  const CFreal* nodePolyValsAtFlxPnts,
                                  const CFreal* nodePolyValsAtSolPnts,
                                  const bool addUpdCoeff)
{    
  // one thread per cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;

  if (cellID < nbCells) 
  { 
    // current kernel data
    KernelData<CFreal> kd (nbCells, states, updateCoeff, rhs, solPntNormals, flxPntNormals, faceDir, nbSolPnts);

    // current flux data
    FluxData<typename SCHEME::MODEL,ORDER> currFd; 

    // initialize flux data
    currFd.initialize();
    
    // physical model
    typename SCHEME::MODEL pmodel(dcop);
    SCHEME fluxScheme(dcof);
    
    PHYSNS pmodelNS(dcopNS,dcopNSConv);    
    
    // current cell data
    CellData cells(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);
    
    // get current cell
    CellData::Itr cell = cells.getItr(cellID);
          
    // initialize constants and vectors
    const CFuint nbFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1)*2*PHYS::DIM;//8;
    
    //const CFuint nbFaceFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1);
   
    const CFuint nbrFaceFlxPnts = (ORDER+1);

    const CFuint totNbrFlxPnts = (ORDER+1)*2*PHYS::DIM;

    const CFuint nbNormals = PHYS::DIM*PHYS::DIM;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntFlx;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSol;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx*PHYS::DIM> flxPntGrads;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx*PHYS::DIM> flxPntGradsAV;
    
    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS*(ORDER+1)*(ORDER+1)*PHYS::DIM> solPntFlx;
    
    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS> avgSol;
    
    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS*PHYS::DIM> avgGrad;
    
    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS*PHYS::DIM> avgGradAV;    
    
    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS> currFlxPntFlx;
    
    CudaEnv::CFVec<CFreal,(ORDER+1)*(ORDER+1)> solEpsilons;    
    
    flxPntFlx = 0.0;
    
    flxPntSol = 0.0;
    
    flxPntGrads = 0.0;
    
    flxPntGradsAV = 0.0;    
    
    solPntFlx = 0.0;

    avgSol = 0.0;
    
    avgGrad = 0.0;
    
    avgGradAV = 0.0;
    
    currFlxPntFlx = 0.0;
    
    const CFreal currVol = cellVolumes[cellID];
    
    solEpsilons = 0.0;

    // loop over flx pnts to extrapolate the states to the flux points
    for (CFuint iSol = 0; iSol < nbSolPnts; ++iSol)
    {   
      // loop over the sol pnts to compute the states and grads in the flx pnts
      for (CFuint iNode = 0; iNode < nbrCornerNodes; ++iNode)
      {
        // get node local index
        const CFuint nodeIdx = neighbNodeIDs[cellID*nbrCornerNodes+iNode];
      
        solEpsilons[iSol] += nodePolyValsAtSolPnts[iSol*nbrCornerNodes+iNode]*nodeEpsilons[nodeIdx]/nbNodeNeighbors[nodeIdx];
      }
      //if (cellID == 0) printf("eps %e\n",solEpsilons[iSol]);
    }
    
    // loop over sol pnts to compute flux
    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);
    
      setFluxData(stateID, cellID, &kd, &currFd, iSolPnt);

      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS*PHYS::DIM> grad(&gradients[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);
      
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS*PHYS::DIM> gradAV(&gradientsAV[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

      CudaEnv::CFVecSlice<CFreal,nbNormals> n(&(kd.solPntNormals[stateID*nbNormals]));

      CudaEnv::CFVecSlice<CFreal,nbNormals> nFd(currFd.getScaledNormal(iSolPnt));
      
      for (CFuint i = 0; i < nbNormals; ++i) 
      {
        nFd[i] = n[i];
      }
          
      for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
      {
        CFreal nJacob2 = 0.0;

        for (CFuint jDir = 0; jDir < PHYS::DIM; ++jDir)
        {
          nJacob2 += n[iDir*PHYS::DIM+jDir]*n[iDir*PHYS::DIM+jDir];
        }
      }
      
      // get the flux
      fluxScheme.prepareComputation(&currFd, &pmodel);

      fluxScheme(&currFd, &pmodel, iSolPnt);
      
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);
            
      for (CFuint iDim = 0; iDim < PHYS::DIM; ++iDim)
      {
        pmodelNS.getUpdateVS()->getFlux(&currState[0],&grad[0],&n[iDim*PHYS::DIM],&solPntFlx[iSolPnt*SCHEME::MODEL::NBEQS*PHYS::DIM+iDim*SCHEME::MODEL::NBEQS]);
        
        for (CFuint iDim2 = 0; iDim2 < PHYS::DIM; ++iDim2)
        {
          for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
          {  
          
            solPntFlx[iSolPnt*SCHEME::MODEL::NBEQS*PHYS::DIM+iDim*SCHEME::MODEL::NBEQS+iEq] += gradAV[iEq*PHYS::DIM+iDim2]*solEpsilons[iSolPnt]*n[iDim*PHYS::DIM+iDim2];
            
            //if (cellID == 0) printf("first iSol: %d, iDir: %d, iEq: %d, gradAV: %e, eps: %e, n: %e\n", iSolPnt, iDim, iEq, gradAV[iEq*PHYS::DIM+iDim2],solEpsilons[iSolPnt],n[iDim*PHYS::DIM+iDim2]);
          }
        }
      }
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      setFluxData(stateID, cellID, &kd, &currFd, iSolPnt);

      // get current vector slice out of rhs
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);

      // Loop over solution pnts to count the factor of all sol pnt polys
      for (CFuint jSolPnt = 0; jSolPnt < nbrSolSolDep; ++jSolPnt)
      { 
        const CFuint jSolIdx = solSolDep[iSolPnt*nbrSolSolDep+jSolPnt]; //(*m_solSolDep)[iSolPnt][jSolPnt];

        // Loop over deriv directions and sum them to compute divergence
        for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
        {
          const CFreal polyCoef = solPolyDerivAtSolPnts[iSolPnt*PHYS::DIM*nbSolPnts+iDir*nbSolPnts+jSolIdx];//(*m_solPolyDerivAtSolPnts)[jSolPnt][iDir][iSolIdx]; 
          
          // Loop over conservative fluxes 
          for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
          {
            // Store divFD in the vector that will be divFC
            res[iEq] -= polyCoef*(currFd.getFlux(jSolIdx, iDir)[iEq] - solPntFlx[jSolIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iDir*SCHEME::MODEL::NBEQS+iEq])*resFactor;

//if (cellID == 11) printf("State: %d, jSol: %d, iDir: %d, var: %d, flx: %f\n",iSolPnt,jSolIdx,iDir,iEq,polyCoef*(solPntFlx[jSolIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iDir*SCHEME::MODEL::NBEQS+iEq])*resFactor);  
	  }
        }
      }
    }

    // extrapolate the fluxes to the flux points
    for (CFuint iFlxPnt = 0; iFlxPnt < nbrFlxPnts; ++iFlxPnt)
    {
      const CFuint dim = flxPntFlxDim[iFlxPnt];

      // loop over sol pnts to compute flux
      for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
      {
        const CFuint solIdx = flxSolDep[iFlxPnt*nbrFlxSolDep + iSolPnt];

        // Loop over conservative fluxes 
        for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
        {
          flxPntFlx[iFlxPnt*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*(currFd.getFlux(solIdx, dim)[iEq] - solPntFlx[solIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+dim*SCHEME::MODEL::NBEQS+iEq]);

          flxPntSol[iFlxPnt*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*states[cell.getStateID(solIdx)*SCHEME::MODEL::NBEQS+iEq];

          for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
          {
            flxPntGrads[iFlxPnt*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*gradients[cell.getStateID(solIdx)*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir];
            flxPntGradsAV[iFlxPnt*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*gradientsAV[cell.getStateID(solIdx)*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir];
          }          
        }
      }
    }

    // set extrapolated states
    for (CFuint iState = 0; iState < nbrFlxPnts; ++iState)
    {
      for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) 
      {
        currFd.getLstate(iState)[iEq] = flxPntSol[iState*PHYS::NBEQS+iEq];
      } 
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      // get current vector slice out of rhs
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);

      // add divhFD to the residual updates
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrSolFlxDep; ++iFlxPnt)
      {
        const CFuint flxIdx = solFlxDep[iSolPnt*nbrSolFlxDep+iFlxPnt];

        // get the divergence of the correction function
        const CFreal divh = corrFctDiv[iSolPnt*nbrFlxPnts+flxIdx];

        // Fill in the corrections
        for (CFuint iVar = 0; iVar < SCHEME::MODEL::NBEQS; ++iVar)
        {
          res[iVar] += flxPntFlx[flxIdx*SCHEME::MODEL::NBEQS+iVar] * divh * resFactor;
//if (cellID == 11 && iVar == 2) printf("State: %d, flx: %d, var: %d, update: %e, flux: %e, divh: %e\n",iSolPnt,flxIdx,iVar,flxPntFlx[flxIdx*SCHEME::MODEL::NBEQS+iVar] * divh, flxPntFlx[flxIdx*SCHEME::MODEL::NBEQS+iVar], divh);  
        }
      }
    }

    // reset flx pnt fluxes  
    flxPntFlx = 0.0;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSolNeighb;
        
    CudaEnv::CFVec<CFreal,nbFlxPntFlx*PHYS::DIM> flxPntGradNeighb;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx*PHYS::DIM> flxPntGradNeighbAV;
    
    flxPntSolNeighb = 0.0;
        
    flxPntGradNeighb = 0.0;
    
    flxPntGradNeighbAV = 0.0;
    
    for (CFuint iFlxPnt = 0; iFlxPnt < nbFlxPntFlx; ++iFlxPnt) 
    {
        flxPntSolNeighb[iFlxPnt] = 0.0;
    }

    // current neighb cell data
    CellData cells2(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);

    for (CFuint iFace = 0; iFace < nbrFaces; ++iFace)
    {
      const CFint neighbCellID = cell.getNeighbCellID(iFace);  

      // get current cell
      CellData::Itr cell2 = cells2.getItr(neighbCellID);
      
      // check if it is a bnd face (if so do nothing): for nbd face the neighbCellID will be -1
      if (neighbCellID != -1)
      {
        CFuint jFaceIdx = 0;
          
        for (CFuint jFace = 0; jFace < nbrFaces; ++jFace)
        {
          if (cell2.getNeighbCellID(jFace) == cellID)
          {
            jFaceIdx = jFace; 
            break;
          }
        }

        CFreal waveSpeedUpd = 0.0;
      
        const CFuint faceID = cell.getNeighbFaceID(iFace);

        const bool isLEFT = (bool) cell.getInnerCellIsLeft(iFace);

      // loop over face flx pnts
      for (CFuint iFlxPnt = 0; iFlxPnt < nbrFaceFlxPnts; ++iFlxPnt)
      { 
        // @TODO check if this also works for non QUADs
        const CFuint flxIdx = faceFlxPntConn[iFace*nbrFaceFlxPnts+iFlxPnt];
        const CFuint jFlxIdx = faceFlxPntConn[jFaceIdx*nbrFaceFlxPnts+nbrFaceFlxPnts-1-iFlxPnt];
        
        
        
        
       
    
    // reset the states in the flx pnts
    CFreal epsL = 0.0;
//    CFreal epsR = 0.0;
    
      //m_cellNodes = m_cells[LEFT]->getNodes();

      for (CFuint iNode = 0; iNode < nbFaceNodes; ++iNode)
      {
        const CFuint faceNodeIdx = faceNeighbNodeIDs[faceID*nbFaceNodes+iNode];
          
	for (CFuint iNodeCell = 0; iNodeCell < nbrCornerNodes; ++iNodeCell)
        {
          const CFuint nodeIdx = neighbNodeIDs[cellID*nbrCornerNodes+iNodeCell];
            
          //if(cellID == 0) printf("faceID: %d, faceNodeID: %d, cellNodeIdD: %d\n",faceID,faceNodeIdx,nodeIdx);
          
	  if (faceNodeIdx == nodeIdx)
	  {
	    // get node local index
            //const CFuint nodeIdx = (*m_cellNodesConn)(m_cells[LEFT]->getID(),iNodeCell);
	    
            epsL += nodePolyValsAtFlxPnts[flxIdx*nbrCornerNodes+iNodeCell]*nodeEpsilons[nodeIdx]/nbNodeNeighbors[nodeIdx];
            
            //if(cellID == 0) printf("node eps: %e, polyVal: %e, nbNeighb: %d\n",nodeEpsilons[nodeIdx],nodePolyValsAtFlxPnts[flxIdx*nbrCornerNodes+iNodeCell],nbNodeNeighbors[nodeIdx]);
	  }
	}
      }
      
//      m_cellNodes = m_cells[RIGHT]->getNodes();
//
//      for (CFuint iNode = 0; iNode < m_faceNodes->size(); ++iNode)
//      {
//	for (CFuint iNodeCell = 0; iNodeCell < m_nbrCornerNodes; ++iNodeCell)
//        {
//	  if ((*m_faceNodes)[iNode]->getLocalID() == (*m_cellNodes)[iNodeCell]->getLocalID())
//	  {
//	    // get node local index
//            const CFuint nodeIdx = (*m_cellNodesConn)(m_cells[RIGHT]->getID(),iNodeCell);
//	    
//            epsR += m_nodePolyValsAtFlxPnts[jFlxIdx][iNodeCell]*m_nodeEpsilons[nodeIdx]/m_nbNodeNeighbors[nodeIdx];
//	  }
//	}
//      }
  
  
  
  
  

        // loop over sol pnts to compute sol at flx pnt
        for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
        {
          const CFuint solIdx = flxSolDep[jFlxIdx*nbrFlxSolDep+iSolPnt]; 

          // Loop over conservative vars 
          for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
          {
            flxPntSolNeighb[flxIdx*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*states[cell2.getStateID(solIdx)*SCHEME::MODEL::NBEQS+iEq];
                        
            for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
            {
              flxPntGradNeighb[flxIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*gradients[cell2.getStateID(solIdx)*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir];
              flxPntGradNeighbAV[flxIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*gradientsAV[cell2.getStateID(solIdx)*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir];
            }
          }
        }

        for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) 
        {
          currFd.getRstate(flxIdx)[iEq] = flxPntSolNeighb[flxIdx*PHYS::NBEQS+iEq];
          
          avgSol[iEq] = 0.5*(flxPntSolNeighb[flxIdx*PHYS::NBEQS+iEq] + flxPntSol[flxIdx*SCHEME::MODEL::NBEQS+iEq]);
          
          for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
          {
            avgGrad[iEq*PHYS::DIM+iDir] = 0.5*(flxPntGradNeighb[flxIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir] + flxPntGrads[flxIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir]);
            avgGradAV[iEq*PHYS::DIM+iDir] = 0.5*(flxPntGradNeighbAV[flxIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir] + flxPntGradsAV[flxIdx*SCHEME::MODEL::NBEQS*PHYS::DIM+iEq*PHYS::DIM+iDir]);
          }
        } 

        CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(&(kd.flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM]));

        CudaEnv::CFVecSlice<CFreal,PHYS::DIM> nFd(currFd.getFlxScaledNormal(flxIdx));

        CFreal faceVecAbsSize2 = 0.0;
        
        for (CFuint i = 0; i < PHYS::DIM; ++i) 
        {
          nFd[i] = n[i];
          
          faceVecAbsSize2 += n[i]*n[i];
        }

        // get the flux
        fluxScheme.prepareComputation(&currFd, &pmodel);

        fluxScheme(&currFd, &pmodel, iFlxPnt, flxIdx, faceIntCoeff[iFlxPnt], isLEFT, waveSpeedUpd);

        // add diff contribution to wvspd upd
        const CFreal mu = pmodelNS.getUpdateVS()->getDynViscosity(&avgSol[0]);
        const CFreal rho = pmodelNS.getUpdateVS()->getDensity(&avgSol[0]);
        
        const CFreal factorPr = 0.72;//min(pmodelNS.getUpdateVS()->getModel().getPrandtl(),1.0);
        
        if (addUpdCoeff)
        {
          waveSpeedUpd += (mu/rho/factorPr+epsL)*faceVecAbsSize2*faceIntCoeff[iFlxPnt]/currVol*cflConvDiffRatio;
        }
        else
        {
          waveSpeedUpd += mu/rho/factorPr*faceVecAbsSize2*faceIntCoeff[iFlxPnt]/currVol*cflConvDiffRatio; 
        }

        pmodelNS.getUpdateVS()->getFlux(&avgSol[0],&avgGrad[0],&n[0],&currFlxPntFlx[0]);
        
        // compute artificial part
        // get epsilon
        const CFreal epsilon = epsL;//0.5*(epsL+epsR);
    
        for (CFuint iDim = 0; iDim < PHYS::DIM; ++iDim)
        {
          for (CFuint iVar = 0; iVar < SCHEME::MODEL::NBEQS; ++iVar)
          {
            currFlxPntFlx[iVar] += epsilon*avgGradAV[iVar*PHYS::DIM+iDim]*n[iDim];
            
            //if(cellID == 0) printf("second flx: %d, var: %d, dim: %d, eps: %e, grad: %e, n: %e\n",iFlxPnt,iVar,iDim,epsilon,avgGradAV[iVar*PHYS::DIM+iDim],n[iDim]*faceDir[cellID*totNbrFlxPnts+flxIdx]); 
          }
        }
        
        // extrapolate the fluxes to the flux points
        for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
        {     
          const CFuint solIdx = flxSolDep[flxIdx*nbrFlxSolDep+iSolPnt];

          // get current state ID
          const CFuint stateID = cell.getStateID(solIdx);

          // get current vector slice out of rhs
          CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> res(&rhs[stateID*SCHEME::MODEL::NBEQS]);   

          // divergence of the correction function
          const CFint currFaceDir = faceDir[cellID*totNbrFlxPnts+flxIdx];
          const CFreal divh = corrFctDiv[solIdx*nbrFlxPnts+flxIdx] * currFaceDir;
          // Fill in the corrections
          for (CFuint iVar = 0; iVar < SCHEME::MODEL::NBEQS; ++iVar)
          {
            res[iVar] -= (currFd.getInterfaceFlux(flxIdx)[iVar] - currFlxPntFlx[iVar]) * divh * resFactor;
//if(cellID == 11 && flxIdx == 1 && iVar == 2) printf("State: %d, flx: %d, var: %d, divh: %e. up: %e\n",solIdx,flxIdx,iVar,divh,currFlxPntFlx[iVar] * divh * resFactor); 
          }
        }
      }

      //CFreal* waveSpeedUpd = currFd.getUpdateCoeff();

      for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
      {  
        // get current state ID
        const CFuint stateID = cell.getStateID(iSolPnt);

        updateCoeff[stateID] += waveSpeedUpd*(2.0*ORDER+1);
      }
 
      //currFd.resetUpdateCoeff();
      }
    }
    
//    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
//      {  
//        // get current state ID
//        const CFuint stateID = cell.getStateID(iSolPnt);
//
//        printf("cellID: %d, stateID: %d, resV: %e\n", cellID, stateID, rhs[stateID*SCHEME::MODEL::NBEQS+2]);
//      }
  }
}
  
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYS, typename PHYSNS, CFuint ORDER>
__global__ void computeGradientsKernel(typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                       typename PHYSNS::DTERM::template DeviceConfigOptions<NOTYPE>* dcopNS,
                                       typename PHYSNS::PTERM::template DeviceConfigOptions<NOTYPE>* dcopNSConv,
                                       const CFuint nbCells,
				       CFreal* states, 
                                       CFreal* gradients,
                                       CFreal* gradientsAV,
                                       CFreal* solPntNormals,
                                       CFreal* flxPntNormals,
                                       CFreal* cellVolumes,
                                       CFreal* volumes,
                                       CFint* faceDir,
                                       CFreal* nodeEpsilons,
                                       CFreal* cellEpsilons,
                                       const CFuint nbSolPnts,
                                       const CFuint nbrFaces,
                                       const CFuint* faceFlxPntConn,
                                       const CFuint* stateIDs,
                                       const CFint* neighbCellIDs,
                                       const CFuint* neighbFaceIDs,
                                       const CFuint* neighbNodeIDs,
                                       const CFuint* innerCellIsLeft,
                                       const CFuint nbrFlxPnts,
                                       const CFuint nbrSolSolDep,
                                       const CFuint* solSolDep,
                                       const CFuint nbrSolFlxDep,
                                       const CFuint* solFlxDep,
                                       const CFuint nbrFlxSolDep,
                                       const CFuint* flxSolDep,
                                       const CFreal* solPolyDerivAtSolPnts,
                                       const CFreal* solPolyValsAtFlxPnts,
                                       const CFuint* flxPntFlxDim,
                                       const CFreal* corrFctDiv,
                                       const CFreal* transformationMatrix,
                                       const CFreal peclet,
                                       const CFreal subcellRes,
                                       const CFreal kappa,
                                       const CFreal s0,
                                       const CFuint monitoredVar,
                                       const CFreal monitoredPhysVar,
                                       const CFuint nbrCornerNodes,
                                       const bool useMax,
                                       const bool flagComputeNbNghb)
{    
  // one thread per cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) 
  { 
    // current kernel data
    //KernelData<CFreal> kd (nbCells, states, updateCoeff, rhs, solPntNormals, flxPntNormals, faceDir, nbSolPnts);

    // physical model
    typename SCHEME::MODEL pmodel(dcop);
    //typename PHYSNS pmodelNS(dcopNS);
    PHYSNS pmodelNS(dcopNS,dcopNSConv);    

    // current cell data
    CellData cells(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);
    
    // get current cell
    CellData::Itr cell = cells.getItr(cellID);
          
    const CFuint nbFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1)*2*PHYS::DIM;//8;
    
    //const CFuint nbFaceFlxPntFlx = SCHEME::MODEL::NBEQS*(ORDER+1);
   
    const CFuint nbrFaceFlxPnts = (ORDER+1);

    const CFuint totNbrFlxPnts = (ORDER+1)*2*PHYS::DIM;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntFlx;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSol;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSolAV;
    
    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS*(ORDER+1)*(ORDER+1)> projStates;

    CudaEnv::CFVec<CFreal,SCHEME::MODEL::NBEQS> stateGradVars;
    //typename MathTypes<CFreal, GPU, SCHEME::MODEL::NBEQS>::VEC stateGradVars;
    
    CudaEnv::CFVec<CFreal,SCHEME::MODEL::DATASIZE> pdata;
    
    flxPntFlx = 0.0;
    
    flxPntSol = 0.0;
    
    flxPntSolAV = 0.0;

    stateGradVars = 0.0;
    
    projStates = 0.0;  
    
    pdata = 0.0;
    
    CFreal currVol = cellVolumes[cellID];
   
    ////////COMPUTE PROJECTED STATES//////////////////////////////////////////////////////////////////////
    if (ORDER != 1)
    {
      for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
      {
        for (CFuint iSol = 0; iSol < nbSolPnts; ++iSol)
        {
          // get current state ID
          const CFuint stateID = cell.getStateID(iSol);
        
          CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);
          
          for (CFuint jSol = 0; jSol < nbSolPnts; ++jSol)
          {
            projStates[jSol*SCHEME::MODEL::NBEQS+iEq] += currState[iEq]*transformationMatrix[jSol*nbSolPnts+iSol];
          }
        }
      }
    }
  else
  {
    for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
    {
      CFreal stateSum = 0.0;
      
      for (CFuint iSol = 0; iSol < nbSolPnts; ++iSol)
      {
        // get current state ID
        const CFuint stateID = cell.getStateID(iSol);
        
        CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);
        
        stateSum += currState[iEq];
      }

      stateSum /= nbSolPnts;

      for (CFuint iSol = 0; iSol < nbSolPnts; ++iSol)
      {
        projStates[iSol*SCHEME::MODEL::NBEQS+iEq] = stateSum;
        
        //if (cellID == 0) printf("projState %d, %d: %e\n",iSol, iEq, stateSum);
      }
    }
  }
    
    
    
  ////////COMPUTE EPS0//////////////////////////////////////////////////////////////////
    
  // compute a cell average characteristic flow speed. Note that a straight average is used, not a weighted one, maybe change this
  CFreal wavespeed = 0.0;

  for (CFuint iSol = 0; iSol < nbSolPnts; ++iSol)
  {    
    // get current state ID
    const CFuint stateID = cell.getStateID(iSol);
        
    CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);
    
    pmodel.getUpdateVS()->computePhysicalData(&currState[0], &pdata[0]);

    wavespeed += pdata[EulerTerm::V] + pdata[EulerTerm::A];
  }
  
  wavespeed /= nbSolPnts;
      
  const CFreal oneOverDim = 1./PHYS::DIM;
  
  const CFreal h = pow(currVol,oneOverDim);

  const CFreal eps0 = max(h*wavespeed*(2.0/peclet - subcellRes/peclet),0.0);
  
  //if (cellID == 0) printf("eps0 %e\n",eps0);
  
  
  
  ////////COMPUTE SMOOTHNESS///////////////////////////////////////////////////////////
  
  CFreal sNum = 0.0;
  
  CFreal sDenom = 0.0;

  if (monitoredPhysVar < SCHEME::MODEL::DATASIZE)
  {
    for (CFuint iSol = 0; iSol < nbSolPnts; ++iSol)
    {
      CFreal stateP = 0.0;
      CFreal diffStatesPPMinOne = 0.0;
      
      // get current state ID
      const CFuint stateID = cell.getStateID(iSol);
        
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);
      
      pmodel.getUpdateVS()->computePhysicalData(&currState[0], &pdata[0]);
      
      stateP = pdata[monitoredPhysVar];
      
      pmodel.getUpdateVS()->computePhysicalData(&projStates[iSol*SCHEME::MODEL::NBEQS], &pdata[0]);

      diffStatesPPMinOne = stateP - pdata[monitoredPhysVar];

      sNum += diffStatesPPMinOne*diffStatesPPMinOne;
      sDenom += stateP*stateP;
    }
  }
  else
  {
    for (CFuint iSol = 0; iSol < nbSolPnts; ++iSol)
    {
      CFreal stateP = 0.0;
      CFreal diffStatesPPMinOne = 0.0;
      
      // get current state ID
      const CFuint stateID = cell.getStateID(iSol);
        
      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);

      stateP = currState[monitoredVar];
      diffStatesPPMinOne = stateP - projStates[iSol*SCHEME::MODEL::NBEQS+monitoredVar];

      sNum += diffStatesPPMinOne*diffStatesPPMinOne;
      sDenom += stateP*stateP;
    }
  }
  
  CFreal smoothness = 0.0;
      
  if (sNum <= 1.0e-10 || sDenom <= 1.0e-10)
  {
    smoothness = -100.0;
  }
  else
  {
    smoothness = log10(sNum/sDenom);
  }
  
  //if (cellID == 0) printf("s %e\n",smoothness);
  
  ///////COMPUTE EPS///////////////////////////////////////////////////////////////////
  
  CFreal eps = 0.0;
  
  if (smoothness > s0 + kappa)
  {
    eps = eps0;
  }
  else if (smoothness > s0 - kappa)
  {
    eps = eps0*0.5*(1.0 + sin(0.5*3.141592653589793238462643383*(smoothness-s0)/kappa));
  }
  
  //if (cellID == 0) printf("eps %e, s %e, s0 %e, k %e\n",eps,smoothness,s0,kappa);

//  if (m_useWallCutOff)
//  {
//    // Get the wall distance
//    DataHandle< CFreal > wallDist = socket_wallDistance.getDataHandle();
//  
//    CFreal centroidDistance = 0.0;
//      
//    for (CFuint iSol = 0; iSol < m_nbrSolPnts; ++iSol)
//    {
//      const CFuint stateID = (*m_cellStates)[iSol]->getLocalID();
//      centroidDistance += wallDist[stateID];
//    }
//    
//    centroidDistance /= m_nbrSolPnts;
//    
//    if (centroidDistance < m_wallCutOff) 
//    {
//      if (centroidDistance < 0.5*m_wallCutOff)
//      {
//        m_epsilon = 0.0; 
//      }
//      else
//      {
//        m_epsilon *= 0.5*(1.0 + sin(0.5*MathTools::MathConsts::CFrealPi()*(centroidDistance-0.75*m_wallCutOff)/(0.25*m_wallCutOff)));
//      }
//    }
//  }
  
  if (eps < 0.0 || eps != eps) 
  {
    eps = 0.0;
  }
  
  
  ////////STORE EPS///////////////////////////////////////////////////////////////////
  
  for (CFuint iNode = 0; iNode < nbrCornerNodes; ++iNode)
  {
    // get node ID
    const CFuint nodeID = neighbNodeIDs[cellID*nbrCornerNodes+iNode];

    if (!useMax) 
    {
      //nodeEpsilons[nodeID] += eps;
      atomicAdd(&nodeEpsilons[nodeID],eps);
      cellEpsilons[cellID] = eps;
    }
    else
    {
      const CFreal maxEps = max(eps, cellEpsilons[cellID]);
      //nodeEpsilons[nodeID] += maxEps;
      atomicAdd(&nodeEpsilons[nodeID],maxEps);
      cellEpsilons[cellID] = maxEps;
    }
  }


    // loop over sol pnts to compute flux
    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      //typename MathTypes<CFreal, GPU, SCHEME::MODEL::NBEQS>::SLICEVEC currState(&states[stateID*SCHEME::MODEL::NBEQS]);

      CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[stateID*SCHEME::MODEL::NBEQS]);

      pmodelNS.getUpdateVS()->setGradientVars(&currState[0],&stateGradVars[0]);

      CudaEnv::CFVecSlice<CFreal,PHYS::DIM*PHYS::DIM> currNormals(&solPntNormals[stateID*PHYS::DIM*PHYS::DIM]);

      // Loop over solution pnts to count the factor of all sol pnt polys
      for (CFuint jSolPnt = 0; jSolPnt < nbrSolSolDep; ++jSolPnt)
      { 
        const CFuint jSolIdx = solSolDep[iSolPnt*nbrSolSolDep+jSolPnt];
        
        // get current j state ID
        const CFuint jStateID = cell.getStateID(jSolIdx);
        
        // get current vector slice out of gradients
        CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> grad(&gradients[jStateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);
        CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> gradAV(&gradientsAV[jStateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

        // Loop over deriv directions and sum them to compute divergence
        for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
        {
          const CFreal polyCoef = solPolyDerivAtSolPnts[jSolIdx*PHYS::DIM*nbSolPnts+iDir*nbSolPnts+iSolPnt];
          
          for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
          {
            for (CFuint jDir = 0; jDir < PHYS::DIM; ++jDir)
            {            
              // Store divFD in the vector that will be divFC
              grad[iEq*PHYS::DIM+jDir] += polyCoef*currNormals[iDir*PHYS::DIM+jDir]*stateGradVars[iEq];//*states[stateID*SCHEME::MODEL::NBEQS+iEq]; 
              gradAV[iEq*PHYS::DIM+jDir] += polyCoef*currNormals[iDir*PHYS::DIM+jDir]*currState[iEq];//*states[stateID*SCHEME::MODEL::NBEQS+iEq]; 

              //if (cellID == 11) printf("after  iSol: %d, iEq: %d, iDir: %d: %f\n", iSolPnt, iEq, jDir, grad[iEq*PHYS::DIM+jDir]); 
	    }
          }
        }
      }
    }

    // extrapolate the fluxes to the flux points
    for (CFuint iFlxPnt = 0; iFlxPnt < nbrFlxPnts; ++iFlxPnt)
    {
      //const CFuint dim = flxPntFlxDim[iFlxPnt];

      // loop over sol pnts to compute flux
      for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
      {
        const CFuint solIdx = flxSolDep[iFlxPnt*nbrFlxSolDep + iSolPnt];

        CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[cell.getStateID(solIdx)*SCHEME::MODEL::NBEQS]);

        pmodelNS.getUpdateVS()->setGradientVars(&currState[0],&stateGradVars[0]);

        // Loop over conservative fluxes 
        for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
        {
          flxPntSol[iFlxPnt*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*stateGradVars[iEq];    
          flxPntSolAV[iFlxPnt*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[iFlxPnt*nbSolPnts+solIdx]*currState[iEq]; 
        }
      }
    }

    // reset flx pnt fluxes  
    flxPntFlx = 0.0;
    
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSolNeighb;
    CudaEnv::CFVec<CFreal,nbFlxPntFlx> flxPntSolNeighbAV;
    
    for (CFuint iFlxPnt = 0; iFlxPnt < nbFlxPntFlx; ++iFlxPnt) 
    {
        flxPntSolNeighb[iFlxPnt] = 0.0;
        flxPntSolNeighbAV[iFlxPnt] = 0.0;
    }

    // current neighb cell data
    CellData cells2(nbCells, stateIDs, neighbCellIDs, neighbFaceIDs, innerCellIsLeft, nbrFaces, nbSolPnts, ORDER);

    for (CFuint iFace = 0; iFace < nbrFaces; ++iFace)
    {
      const CFint neighbCellID = cell.getNeighbCellID(iFace);  

      // get current cell
      CellData::Itr cell2 = cells2.getItr(neighbCellID);
      
      if (neighbCellID != -1)
      {
        CFuint jFaceIdx = 0;

        for (CFuint jFace = 0; jFace < nbrFaces; ++jFace)
        {
          if (cell2.getNeighbCellID(jFace) == cellID)
          {
            jFaceIdx = jFace; 
            break;
          }
        }

        const CFuint faceID = cell.getNeighbFaceID(iFace);

        const bool isLEFT = (bool) cell.getInnerCellIsLeft(iFace);

        // loop over face flx pnts
        for (CFuint iFlxPnt = 0; iFlxPnt < nbrFaceFlxPnts; ++iFlxPnt)
        { 
          // @TODO check if this also works for non QUADs
          const CFuint flxIdx = faceFlxPntConn[iFace*nbrFaceFlxPnts+iFlxPnt];
          const CFuint jFlxIdx = faceFlxPntConn[jFaceIdx*nbrFaceFlxPnts+nbrFaceFlxPnts-1-iFlxPnt];

          const CFreal dirFactor = faceDir[cellID*totNbrFlxPnts+flxIdx];

          // loop over sol pnts to compute sol at flx pnt
          for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
          {
            const CFuint solIdx = flxSolDep[jFlxIdx*nbrFlxSolDep+iSolPnt]; 

            CudaEnv::CFVecSlice<CFreal,SCHEME::MODEL::NBEQS> currState(&states[cell2.getStateID(solIdx)*SCHEME::MODEL::NBEQS]);

            pmodelNS.getUpdateVS()->setGradientVars(&currState[0],&stateGradVars[0]);

            // Loop over conservative vars 
            for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
            {
              flxPntSolNeighb[flxIdx*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*stateGradVars[iEq];
              flxPntSolNeighbAV[flxIdx*SCHEME::MODEL::NBEQS+iEq] += solPolyValsAtFlxPnts[jFlxIdx*nbSolPnts+solIdx]*currState[iEq];
            }
          }

          // extrapolate the fluxes to the flux points
          for (CFuint iSolPnt = 0; iSolPnt < nbrFlxSolDep; ++iSolPnt)
          {     
            const CFuint solIdx = flxSolDep[flxIdx*nbrFlxSolDep+iSolPnt];

            // get current state ID
            const CFuint stateID = cell.getStateID(solIdx); 

            // get current vector slice out of gradients
            CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> grad(&gradients[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);
            CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> gradAV(&gradientsAV[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

            // divergence of the correction function
            const CFreal divh = corrFctDiv[solIdx*nbrFlxPnts+flxIdx];
           
            for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
            {
              const CFreal corrFactor = 0.5*(flxPntSolNeighb[flxIdx*SCHEME::MODEL::NBEQS+iEq]-flxPntSol[flxIdx*SCHEME::MODEL::NBEQS+iEq]);
              const CFreal corrFactorAV = 0.5*(flxPntSolNeighbAV[flxIdx*SCHEME::MODEL::NBEQS+iEq]-flxPntSolAV[flxIdx*SCHEME::MODEL::NBEQS+iEq]);

              // Loop over deriv directions and sum them to compute divergence
              for (CFuint iDir = 0; iDir < PHYS::DIM; ++iDir)
              {
                grad[iEq*PHYS::DIM+iDir] += divh*corrFactor*flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM+iDir]*dirFactor; 
                gradAV[iEq*PHYS::DIM+iDir] += divh*corrFactorAV*flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM+iDir]*dirFactor; 

//              if (cellID == 11) printf("iSol: %d, iEq: %d, iFlx %d, iDir: %d: %e\n", solIdx, iEq, flxIdx, iDir,
//                      divh*corrFactor*flxPntNormals[faceID*nbrFaceFlxPnts*PHYS::DIM+iFlxPnt*PHYS::DIM+iDir]*dirFactor);  
	      }
            }
          }
        }
      }
    }

    for (CFuint iSolPnt = 0; iSolPnt < nbSolPnts; ++iSolPnt)
    {  
      // get current state ID
      const CFuint stateID = cell.getStateID(iSolPnt);

      // get current vector slice out of gradients
      CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> grad(&gradients[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);
      CudaEnv::CFVecSlice<CFreal,PHYS::DIM*SCHEME::MODEL::NBEQS> gradAV(&gradientsAV[stateID*SCHEME::MODEL::NBEQS*PHYS::DIM]);

      const CFreal invacob = 1.0/volumes[stateID];

      for (CFuint jDir = 0; jDir < PHYS::DIM; ++jDir)
      {
        // Loop over conservative fluxes 
        for (CFuint iEq = 0; iEq < SCHEME::MODEL::NBEQS; ++iEq)
        {
          grad[iEq*PHYS::DIM+jDir] *= invacob;
          gradAV[iEq*PHYS::DIM+jDir] *= invacob;
	}
      }
    }
  }
}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, typename PHYSICSNS, CFuint ORDER, CFuint NB_BLOCK_THREADS>
void ConvDiffLLAVRHSFluxReconstructionCUDA<SCHEME,PHYSICS,PHYSICSNS,ORDER,NB_BLOCK_THREADS>::execute()
{
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "ConvDiffLLAVRHSFluxReconstructionCUDA::execute() START\n");
  
  // get the elementTypeData
  SafePtr< vector<ElementTypeData> > elemType = MeshDataStack::getActive()->getElementTypeData();

  // get InnerCells TopologicalRegionSet
  SafePtr<TopologicalRegionSet> cells = MeshDataStack::getActive()->getTrs("InnerCells");

  // get the geodata of the geometric entity builder and set the TRS
  StdTrsGeoBuilder::GeoData& geoDataCell = m_cellBuilder->getDataGE();
  geoDataCell.trs = cells;
  
  // get InnerFaces TopologicalRegionSet
  SafePtr<TopologicalRegionSet> faces = MeshDataStack::getActive()->getTrs("InnerFaces");

  // get the face start indexes
  vector< CFuint >& innerFacesStartIdxs = getMethodData().getInnerFacesStartIdxs();

  // get number of face orientations
  const CFuint nbrFaceOrients = innerFacesStartIdxs.size()-1;

  // get the geodata of the face builder and set the TRSs
  FaceToCellGEBuilder::GeoData& geoDataFace = m_faceBuilder->getDataGE();
  geoDataFace.cellsTRS = cells;
  geoDataFace.facesTRS = faces;
  geoDataFace.isBoundary = false;
  
  // loop over element types, for the moment there should only be one
  const CFuint nbrElemTypes = elemType->size();
  cf_assert(nbrElemTypes == 1);
  
  // get start and end indexes for this type of element
  cf_assert((*elemType)[0].getStartIdx() == 0);
  const CFuint nbCells   = (*elemType)[0].getEndIdx();
  cf_assert(nbCells > 0);
  
  //initializeComputationRHS();

  const CFuint nbStates = socket_states.getDataHandle().size();
  cf_assert(nbStates > 0);

  CFLog(VERBOSE, "nbCells: " << nbCells << ", nbStates: " << nbStates << "\n");

  DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = socket_rhs.getDataHandle(); 
  DataHandle<CFreal> solPntNormals = socket_solPntNormals.getDataHandle(); 
  DataHandle<CFreal> flxPntNormals = socket_flxPntNormals.getDataHandle(); 
  DataHandle<CFint> faceDir = socket_faceDir.getDataHandle(); 
  DataHandle<CFreal> gradients = socket_gradientsCUDA.getDataHandle();
  DataHandle<CFreal> gradientsAV = socket_gradientsAVCUDA.getDataHandle();
  DataHandle<CFreal> volumes = socket_volumes.getDataHandle();
  DataHandle<CFreal> cellVolumes = socket_cellVolumes.getDataHandle();
 

  SafePtr<SCHEME> lf  = getMethodData().getRiemannFlux().d_castTo<SCHEME>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();

  SafePtr<typename PHYSICSNS::DTERM> physNS = PhysicalModelStack::getActive()->getImplementor()->
    getDiffusiveTerm().d_castTo<typename PHYSICSNS::DTERM>();

  SafePtr<typename PHYSICSNS::PTERM> physNSConv = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICSNS::PTERM>();
  
#ifdef CF_HAVE_CUDA
  typedef typename SCHEME::template DeviceFunc<GPU, PHYSICS, ORDER> FluxScheme;  
#else
  typedef typename SCHEME::template DeviceFunc<CPU, PHYSICS, ORDER> FluxScheme;
#endif 
  
  // get current iteration
  const CFuint iter = SubSystemStatusStack::getActive()->getNbIter();
  
  // check if LLAV should be frozen
  m_useMax = iter > m_freezeLimiterIter;
  
  if (m_onGPU) 
  {
#ifdef CF_HAVE_CUDA

    //CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
    //timer.start();
      
    // copy of data that change at every iteration
    socket_states.getDataHandle().getGlobalArray()->put(); 
    socket_gradientsCUDA.getDataHandle().getLocalArray()->put();
    socket_gradientsAVCUDA.getDataHandle().getLocalArray()->put();
    socket_rhs.getDataHandle().getLocalArray()->put(); 
    socket_updateCoeff.getDataHandle().getLocalArray()->put();
    
    //CFLog(VERBOSE, "nb normals: " << socket_solPntNormals.getDataHandle().size() << ", n0: " << socket_solPntNormals.getDataHandle()[0] << "\n");

    socket_faceDir.getDataHandle().getLocalArray()->put();
    socket_solPntNormals.getDataHandle().getLocalArray()->put();
    socket_flxPntNormals.getDataHandle().getLocalArray()->put();
    socket_volumes.getDataHandle().getLocalArray()->put();
    socket_cellVolumes.getDataHandle().getLocalArray()->put();

    DataHandle<Framework::State*, Framework::GLOBAL > statesI = socket_states.getDataHandle();
     
    //CFLog(VERBOSE, "ConvDiffLLAVRHSFluxReconstructionCUDA::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    //timer.start();
    
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);
    ConfigOptionPtr<typename PHYSICSNS::DTERM, NOTYPE, GPU> dcopNS(physNS);
    ConfigOptionPtr<typename PHYSICSNS::PTERM, NOTYPE, GPU> dcopNSConv(physNSConv);

    const CFuint nThreads = m_nbCellsPerBlock;//512; //CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    const CFuint blocksPerGrid = ceil(nbCells*1.0/nThreads); //CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    
    CFLog(VERBOSE, "blocksPerGrid: " << blocksPerGrid << ", threads: " << nThreads << "\n");

    // boolean telling whether there is a diffusive term
    const bool hasDiffTerm = getMethodData().hasDiffTerm() || getMethodData().hasArtificialViscosity();

//CFuint megabytesToUse = 24;
//size_t newHeapSize = 1024 * 1000 * megabytesToUse;
//hipDeviceSetLimit(hipLimitMallocHeapSize, newHeapSize);
//printf("Adjusted heap size to be %d\n",(int) newHeapSize);

    //dim3 blocks(m_nbBlocksPerGridX, m_nbBlocksPerGridY);
    
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeGradientsKernel"), hipFuncCachePreferL1);

    // get residual factor
    const CFreal resFactor = getMethodData().getResFactor();
    
    m_nodeEpsilons = 0.0;
    
    m_nodeEpsilons.put();
    m_cellEpsilons.put();
    
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>("computeFluxKernel"), hipFuncCachePreferL1);

    // if there is a diffusive term, compute the gradients
//    if (hasDiffTerm)
//    {
        CFLog(VERBOSE, "grad kernel\n");
      computeGradientsKernel<FluxScheme,PHYSICS,PHYSICSNS,ORDER> <<<blocksPerGrid,nThreads>>>(
                                       dcop.getPtr(),
                                       dcopNS.getPtr(),
                                       dcopNSConv.getPtr(),
                                       nbCells,
				       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
                                       gradients.getLocalArray()->ptrDev(), 
                                       gradientsAV.getLocalArray()->ptrDev(), 
                                       solPntNormals.getLocalArray()->ptrDev(),
                                       flxPntNormals.getLocalArray()->ptrDev(),
                                       cellVolumes.getLocalArray()->ptrDev(),
                                       volumes.getLocalArray()->ptrDev(),
                                       faceDir.getLocalArray()->ptrDev(),
                                       m_nodeEpsilons.ptrDev(),
                                       m_cellEpsilons.ptrDev(),
                                       m_nbrSolPnts,
                                       4,
                                       m_faceFlxPntConn2.ptrDev(),
                                       m_stateIDs.ptrDev(),
                                       m_neighbCellIDs.ptrDev(),
                                       m_neighbFaceIDs.ptrDev(),
                                       m_neighbNodeIDs.ptrDev(),
                                       m_innerCellIsLeft.ptrDev(),
                                       m_nbrFlxPnts,
                                       m_nbrSolSolDep,
                                       m_solSolDep2.ptrDev(),
                                       m_nbrFlxDep,
                                       m_solFlxDep2.ptrDev(),
                                       m_nbrSolDep,
                                       m_flxSolDep2.ptrDev(),
                                       m_solPolyDerivAtSolPnts2.ptrDev(),
                                       m_solPolyValsAtFlxPnts2.ptrDev(),
                                       m_flxPntFlxDim2.ptrDev(),
                                       m_corrFctDiv2.ptrDev(),
                                       m_transformationMatrix2.ptrDev(),
                                       m_peclet,
                                       m_subcellRes,
                                       m_kappa,
                                       m_s0,
                                       m_monitoredVar,
                                       m_monitoredPhysVar,
                                       m_nbrCornerNodes,
                                       m_useMax,
                                       m_flagComputeNbNghb);
//    }
        CFLog(VERBOSE, "sol kernel\n");

    m_flagComputeNbNghb = false;
        
    // compute the convective flux in each cell
    computeStateLocalRHSKernel<FluxScheme,PHYSICS,PHYSICSNS,ORDER> <<<blocksPerGrid,nThreads>>> 
      (dcof.getPtr(),
       dcop.getPtr(),
       dcopNS.getPtr(),
       dcopNSConv.getPtr(),
       nbCells,
       resFactor,
       socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       gradients.getLocalArray()->ptrDev(),
       gradientsAV.getLocalArray()->ptrDev(), 
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       solPntNormals.getLocalArray()->ptrDev(),
       flxPntNormals.getLocalArray()->ptrDev(),
       cellVolumes.getLocalArray()->ptrDev(),
       faceDir.getLocalArray()->ptrDev(),
       m_nbrSolPnts,
       4,
       m_faceFlxPntConn2.ptrDev(),
       m_stateIDs.ptrDev(),
       m_neighbCellIDs.ptrDev(),
       m_neighbFaceIDs.ptrDev(),
       m_innerCellIsLeft.ptrDev(),
       m_nbrFlxPnts,
       m_nbrSolSolDep,
       m_solSolDep2.ptrDev(),
       m_nbrFlxDep,
       m_solFlxDep2.ptrDev(),
       m_nbrSolDep,
       m_flxSolDep2.ptrDev(),
       m_solPolyDerivAtSolPnts2.ptrDev(),
       m_solPolyValsAtFlxPnts2.ptrDev(),
       m_flxPntFlxDim2.ptrDev(),
       m_corrFctDiv2.ptrDev(),
       m_faceIntegrationCoefs2.ptrDev(),
       m_cflConvDiffRatio,
       m_nbNodeNeighbors.ptrDev(),
       m_nodeEpsilons.ptrDev(),
       m_nbrCornerNodes,
       m_neighbNodeIDs.ptrDev(),
       m_faceNeighbNodeIDs.ptrDev(),
       m_nbFaceNodes,
       m_nodePolyValsAtFlxPnts2.ptrDev(),
       m_nodePolyValsAtSolPnts2.ptrDev(),
       m_addUpdCoeff);
    
   
    hipDeviceSynchronize();
    
    
    //for (CFuint i = 0; i < m_solPolyValsAtFlxPnts2.size(); ++i) {CFLog(INFO, "thing: " << m_solPolyValsAtFlxPnts2[i] << "\n");}
    
    //CFLog(INFO, "After Kernel, size: " << socket_states.getDataHandle().size() << "\n");
    
    //CFLog(VERBOSE, "ConvDiffLLAVRHSFluxReconstructionCUDA::execute() => computeFluxKernel took " << timer.elapsed() << " s\n");
    
    //for (CFuint i = 0; i < rhs.size(); ++i) {CFLog(INFO, "res before: " << rhs[i] << "\n");}
    
    //RealVector rhsB;
    //rhsB.resize(rhs.size());
    //for (CFuint i = 0; i < rhs.size(); ++i) {rhsB[i] = rhs[i];}
    
    //timer.start();
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    gradients.getLocalArray()->get();
    gradientsAV.getLocalArray()->get();
    
    //m_nodeEpsilons.get();
    m_cellEpsilons.get();
    
    //for (CFuint i = 0; i < rhs.size(); ++i) {CFLog(INFO, "res after: " << rhs[i]-rhsB[i] << "\n");}
    //CFLog(VERBOSE, "ConvDiffLLAVRHSFluxReconstructionCUDA::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");
    //CFLog(INFO, "resSize: " << rhs.size() << "\n");
    //for (CFuint i = 0; i < rhs.size(); ++i)
    //{
      //if (abs(rhs[i]) > 1.0e-10) CFLog(INFO, "res " << i << ": " << rhs[i] << "\n");
    //}

  #endif
  }
  else 
  {
  // boolean telling whether there is a diffusive term
  const bool hasDiffTerm = getMethodData().hasDiffTerm() || getMethodData().hasArtificialViscosity();

  // loop over element types, for the moment there should only be one
  for (m_iElemType = 0; m_iElemType < nbrElemTypes; ++m_iElemType)
  {
    // get start and end indexes for this type of element
    const CFuint startIdx = (*elemType)[m_iElemType].getStartIdx();
    const CFuint endIdx   = (*elemType)[m_iElemType].getEndIdx();

    // loop over cells
    for (CFuint elemIdx = startIdx; elemIdx < endIdx; ++elemIdx)
    {
      // build the GeometricEntity
      geoDataCell.idx = elemIdx;
      m_cell = m_cellBuilder->buildGE();

      // get the states in this cell
      m_cellStates = m_cell->getStates();
      
      // if the states in the cell are parallel updatable or the gradients need to be computed, set the cell data
      if ((*m_cellStates)[0]->isParUpdatable() || hasDiffTerm)
      {
	// set the cell data
	setCellData();
      }
      
      // if the states in the cell are parallel updatable, compute the divergence of the discontinuous flx (-divFD+divhFD)
      if ((*m_cellStates)[0]->isParUpdatable())
      {
	// compute the divergence of the discontinuous flux (-divFD+divhFD)
	computeDivDiscontFlx(m_divContFlx);
      
	// update RHS
        updateRHS();
      } 
      
      // if there is a diffusive term, compute the gradients
      if (hasDiffTerm)
      {
	computeGradients();
      }
      
      // print out the residual updates for debugging
      if(m_cell->getID() == 35) //true) //
      {
	CFLog(VERBOSE, "ID  = " << (*m_cellStates)[0]->getLocalID() << "\n");
        CFLog(VERBOSE, "coords  = " << (*m_cellStates)[0]->getCoordinates() << "\n");
        CFLog(VERBOSE, "UpdateTotal = \n");
        // get the datahandle of the rhs
        DataHandle< CFreal > rhs = socket_rhs.getDataHandle();
        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
          CFuint resID = m_nbrEqs*( (*m_cellStates)[iState]->getLocalID() );
          for (CFuint iVar = 0; iVar < m_nbrEqs; ++iVar)
          {
            CFLog(VERBOSE, "" << rhs[resID+iVar] << " ");
          }
          CFLog(VERBOSE,"\n");
          DataHandle<CFreal> updateCoeff = socket_updateCoeff.getDataHandle();
          CFLog(VERBOSE, "UpdateCoeff: " << updateCoeff[(*m_cellStates)[iState]->getLocalID()] << "\n");
	  CFLog(VERBOSE, "state " << iState << ": " << *(((*m_cellStates)[iState])->getData()) << "\n");
        }
      }
      
      if(m_cell->getID() == 35 && hasDiffTerm)
      {
	// get the gradients
        DataHandle< vector< RealVector > > gradients = socket_gradients.getDataHandle();

        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
	  CFuint solID = ((*m_cellStates)[iState])->getLocalID();
          for (CFuint iGrad = 0; iGrad < m_nbrEqs; ++iGrad)
          {
	    CFLog(VERBOSE, "total gradient " << iGrad << " of  " << iState << ": " << gradients[solID][iGrad] << "\n");
          } 
        }
        for (CFuint iState = 0; iState < m_nbrSolPnts; ++iState)
        {
	  CFLog(VERBOSE, "state " << iState << ": " << *(((*m_cellStates)[iState])->getData()) << "\n");
	}
      }
      
      //release the GeometricEntity
      m_cellBuilder->releaseGE();
    }
  }
  
  //// Loop over faces to calculate fluxes and interface fluxes in the flux points
  
  // loop over different orientations
  for (m_orient = 0; m_orient < nbrFaceOrients; ++m_orient)
  {
    CFLog(VERBOSE, "Orient = " << m_orient << "\n");
    // start and stop index of the faces with this orientation
    const CFuint faceStartIdx = innerFacesStartIdxs[m_orient  ];
    const CFuint faceStopIdx  = innerFacesStartIdxs[m_orient+1];

    // loop over faces with this orientation
    for (CFuint faceID = faceStartIdx; faceID < faceStopIdx; ++faceID)
    {
      // build the face GeometricEntity
      geoDataFace.idx = faceID;
      m_face = m_faceBuilder->buildGE();

      // get the neighbouring cells
      m_cells[LEFT ] = m_face->getNeighborGeo(LEFT );
      m_cells[RIGHT] = m_face->getNeighborGeo(RIGHT);

      // get the states in the neighbouring cells
      m_states[LEFT ] = m_cells[LEFT ]->getStates();
      m_states[RIGHT] = m_cells[RIGHT]->getStates();

      // if one of the neighbouring cells is parallel updatable or if the gradients have to be computed, set the bnd face data
      if ((*m_states[LEFT ])[0]->isParUpdatable() || (*m_states[RIGHT])[0]->isParUpdatable() || hasDiffTerm)
      {
	// set the bnd face data
        setFaceData(m_face->getID());//faceID

	// compute the states in the flx pnts
        computeFlxPntStates();

	// compute the interface flux
	computeInterfaceFlxCorrection();
          
	// compute the wave speed updates
        computeWaveSpeedUpdates(m_waveSpeedUpd);

        // update the wave speed
        updateWaveSpeed();
      }
	
	// if one of the neighbouring cells is parallel updatable, compute the correction flux
      if ((*m_states[LEFT ])[0]->isParUpdatable() || (*m_states[RIGHT])[0]->isParUpdatable())
      {
	// compute the correction for the left neighbour
	computeCorrection(LEFT, m_divContFlxL);
	
	// compute the correction for the right neighbour
	computeCorrection(RIGHT, m_divContFlxR);
	
	// update RHS
	updateRHSBothSides();
      }
      
      // if there is a diffusive term, compute the gradients
      if (hasDiffTerm)
      {
	// compute the face correction term of the corrected gradients
        computeGradientFaceCorrections();
      }

      // release the GeometricEntity
      m_faceBuilder->releaseGE();
    }
  }

    //DataHandle< CFreal > rhs = socket_rhs.getDataHandle();

    //for (CFuint i = 0; i < rhs.size(); ++i)
    //{
      //if (abs(rhs[i]) > 1.0e-10) CFLog(INFO, "res " << i << ": " << rhs[i] << "\n");
    //}
  }
  
  //finalizeComputationRHS();
  
  CFLog(VERBOSE, "ConvDiffLLAVRHSFluxReconstructionCUDA::execute() END\n");
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FluxReconstructionMethod

} // namespace COOLFluiD
